#include "hip/hip_runtime.h"
% % writefile vectorAdd.cu

#include <math.h>
#include <stdio.h>

        /*
         * CUDA Kernel Device Code
         *
         */

        __global__ void
        vectorAdd(const float *A, const float *B, float *S, int nElem)
{

    int tid_b = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid_b < nElem)
    {
        S[tid_b] = A[tid_b] + B[tid_b];
    }
}

__global__ void vectorProd(const float *A, const float *B, float *P, int nElem)
{

    int tid_b = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid_b < nElem)
    {
        P[tid_b] = A[tid_b] * B[tid_b];
    }
}

/*
 * Host Main Routine
 *
 */
int main(int argc, char *argv[])
{
    int i;
    int bSize = 256;
    int nElem = 10000;
    int nBlock = (nElem + bSize - 1) / bSize;

    size_t size = nElem * sizeof(float);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector S
    float *h_S = (float *)malloc(size);

    float *h_P = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_S == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (i = 0; i < nElem; ++i)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Set the GPU
    hipSetDevice(0);

    // Allocate the device input vector A
    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);

    // Allocate the device input vector B
    float *d_B = NULL;
    hipMalloc((void **)&d_B, size);

    // Allocate the device output vector S
    float *d_S = NULL;
    hipMalloc((void **)&d_S, size);

    float *d_P = NULL;
    hipMalloc((void **)&d_P, size);

    // Copy the host input vectors A and B to the device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    printf("[Vector Addition of %d Elements]\n", nElem);

    // Define the Grid and Block dimensions

    dim3 myGrid(nBlock);
    dim3 myBlock(bSize);

    // Launch the Vector Add CUDA Kernel
    printf("CUDA kernel launch with %d blocks of %d threads\n", nBlock, nElem);
    vectorAdd<<<myGrid, myBlock>>>(d_A, d_B, d_S, nElem);

    vectorProd<<<myGrid, myBlock>>>(d_A, d_B, d_P, nElem);

    // Copy the device result vector S to the host memory
    hipMemcpy(h_S, d_S, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

    // Verify that the resulting vector is correct
    for (i = 0; i < nElem; ++i)
    {
        if (fabs((h_A[i] + h_B[i]) - h_S[i]) > 1e-5)
        {
            fprintf(stderr, "Result Suma verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }

        if (fabs((h_A[i] * h_B[i]) - h_P[i]) > 1e-5)
        {
            fprintf(stderr, "Result Prod verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_S);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_S);

    printf("Done\n");
    return (EXIT_SUCCESS);
}
