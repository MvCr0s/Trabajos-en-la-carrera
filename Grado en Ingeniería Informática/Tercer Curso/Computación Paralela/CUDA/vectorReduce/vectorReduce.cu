#include "hip/hip_runtime.h"
%%writefile vectorReduce.cu

/*
 * This example shows how to compute the reduction of the elements of a vector.
 *
 * Also, it shows how to measure the performance of block of threads of a kernel
 * accurately. Blocks are executed in parallel and out of order. Since there's no
 * synchronization mechanism between blocks, we measure the clock once for each block.
 *
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <math.h>

/*
 * vectorReduce
 *
 * This kernel computes a standard parallel reduction and evaluates the
 * time it takes to do that for each block. The timing results are stored in device memory.
 * 
 */
__global__ void time_and_reduce(float *vector_d, float *reduce_d, clock_t *times_d, int n)
{
	extern __shared__ float sdata[];
	
	// local thread ID (in block)
	int tidb =threadIdx.x; // COMPLETAR...
	
    // global thread (ID in grid)
	int tidg =blockIdx.x * blockDim.x + tidb; // COMPLETAR...
	
	// record the initial time for each block
	if (tidb == 0) {
		times_d[blockIdx.x] = clock();
	}
	
	// move data from global to shared memory
	// COMPLETAR...
	if (tidg < n) {
        sdata[tidb] = vector_d[tidg];
    } else {
        sdata[tidb] = 0.0f;
    }

    __syncthreads();
	
	// perform reduction in shared memory
	for (int s = blockDim.x/2; s > 0; s >>= 1) {
		if (tidb < s) {
			sdata[tidb] += sdata[tidb + s];
		}
		 __syncthreads();
	}
	
	// write result for this block to global memory
	if (tidb == 0) {
		atomicAdd(reduce_d, sdata[0]);
        times_d[blockIdx.x + gridDim.x] = clock() - times_d[blockIdx.x];// COMPLETAR... (vectores reduce_d y times)
	}
}

/*
 * Host main routine
 *
 */
int main(int argc, char **argv)
{
	// default parameter values
	int n = 1024, bsx = 32;
	
	size_t nBytes = n * sizeof(float);
	
	clock_t *clocks_h = NULL;
	clock_t *clocks_d = NULL;
	
	float elapsed_time = .0;
	float *vector_h, *reduce_h;	// host data
    float *vector_d, *reduce_d;	// device data
	
	// set the GPU to use
	hipSetDevice(0);// COMPLETAR...
	
	// total number of thread blocks
	int nblocks = (n + bsx - 1) / bsx;// COMPLETAR...

	// set kernel launch configuration
    dim3 grid( nblocks );// COMPLETAR... 
    dim3 block( bsx );
	
    // allocate host memory
    vector_h = (float *) malloc(nBytes);
    clocks_h = (clock_t *) malloc(2 * nblocks * sizeof(clock_t));// COMPLETAR...
    reduce_h = (float *) malloc(sizeof(float));// COMPLETAR...
	
	float acum = .0;
	// initialize host memory
    for(int i = 0; i < n; i++) {
        vector_h[i] = (float) 1;
		acum += 1.0;
	}
	
    // allocate device memory
    hipMalloc((void **) &vector_d, nBytes );// COMPLETAR...
    hipMalloc((void **) &reduce_d, sizeof(float) );// COMPLETAR...
	hipMalloc((void **) &clocks_d, 2 * nblocks * sizeof(clock_t));// COMPLETAR...
	hipMemset(reduce_d, 0, sizeof(float));

	// create cuda events
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop );
	
	// insert stream 0 in start event
	hipEventRecord(start, 0);
	
    // copy data from host memory to device memory
    hipMemcpy(vector_d, vector_h, nBytes, hipMemcpyHostToDevice);
    
    // execute the kernel 
    printf("---> Running configuration: grid of %d blocks of %d threads (TOTAL: %d threads)\n", nblocks, bsx, nblocks * bsx );
    time_and_reduce<<< grid, block,  bsx * sizeof(float) >>>(vector_d, reduce_d, clocks_d, n);// COMPLETAR... 

    // copy data from device memory to host memory
	hipMemcpy(clocks_h, clocks_d, 2 * nblocks * sizeof(clock_t), hipMemcpyDeviceToHost);
    hipMemcpy(reduce_h, reduce_d, sizeof(float), hipMemcpyDeviceToHost);// COMPLETAR...
	
	// insert stream 0 in stop event
	hipEventRecord(stop, 0);

    // using events to calculate the execution time        
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("---> Time spent executing by the GPU: %.2f\n", elapsed_time);
	
	long double avgElapsedClocks = 0;
    for (int i = 0; i < nblocks; i++) {
		avgElapsedClocks += (long double) clocks_h[i];
    }
    avgElapsedClocks = avgElapsedClocks / nblocks;
    printf("Average Clocks/Block = %Lf\n", avgElapsedClocks);

	// check the output for correctness
	for(int i = 1; i < nblocks; i++) { reduce_h[0] += reduce_h[i]; }
	assert(reduce_h[0] == (float) acum);


	// destroy events
	hipEventDestroy(start);
	hipEventDestroy(stop );

    // free host memory
    free(vector_h);
	free(reduce_h);
	free(clocks_h);
	
	// free device memory
    hipFree((void *) vector_d);
    hipFree((void *) reduce_d);
	hipFree((void *) clocks_d);
	
    printf("\nTest PASSED\n");
	exit(EXIT_SUCCESS);
}
