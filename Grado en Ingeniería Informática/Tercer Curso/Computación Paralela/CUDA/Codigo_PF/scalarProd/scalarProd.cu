#include "hip/hip_runtime.h"
%%writefile scalarProd.cu

/*
 * This file contains both device and host code to calculate the
 * scalar product of two vectors of N elements.
 * 
 */

#include <stdio.h>

#define N 1024
#define SEGMENT_SIZE 64

///////////////////////////////////////////////////////////////////////////////
//
// Computes the scalar product of two vectors of N elements on GPU.
//
///////////////////////////////////////////////////////////////////////////////
__global__ void scalarProd(float *C, const float *A, const float *B, int nElem) {

	// COMPLETAR...
}

/////////////////////////////////////////////////////////////////
//
// Computes a standard parallel reduction on GPU.
//
/////////////////////////////////////////////////////////////////
__global__ void vectorReduce(float *R, const float *C, int nElem)
{
	// Array in Shared Memory
    extern __shared__ float sdata[];
	
	// COMPLETAR...
}

// -----------------------------------------------
// Host Utility Routines
// -----------------------------------------------
float scalarProd_CPU(float *A, float *B, int nElem)
{
	float suma = 0.0f;	
	for (int i = 0; i < nElem; i++) {
		suma += A[i] * B[i];
	}
	return suma;
}

float randFloat(float low, float high) {
	float t = (float) rand() / (float) RAND_MAX;
	return (1.0f - t) * low + (t * high);
}

// ------------
// Main Program
// ------------
int main( void ) {

	// Array Elements
	int n_elem = N;
	
	// Block Dimension
	int block_dim = SEGMENT_SIZE;
	
	// Number of Blocks
	int n_block = ( n_elem % block_dim == 0 ) // COMPLETAR...
	
	// Execution Configuration Parameters
	dim3 blocks ( // COMPLETAR... );
	dim3 threads( // COMPLETAR... );
	
	// Size (in bytes) Required to Store the Matrix
	size_t n_bytes = (n_elem * sizeof(float));
	
	// Allocate Host Memory
	float *h_A = (float *) malloc( // COMPLETAR... );
	float *h_B = (float *) malloc( // COMPLETAR... );
	float *h_R = (float *) malloc( // COMPLETAR... );
		
	// Initialize Host Data
	srand(123);
	
	// Generating input data on CPU
	for (int i=0; i < n_elem; i++) {
		h_A[i] = randFloat(0.0f, 1.0f);
		h_B[i] = randFloat(0.0f, 1.0f);
	}
	
	// Compute Reference CPU Solution
	float result_cpu = scalarProd_CPU(h_A, h_B, n_elem);
	
	// CUDA Events
	hipEvent_t start, stop;
	
	// Allocate Device Memory
	float *d_A, *d_B, *d_C, *d_R;
	hipMalloc((void **)&d_A, // COMPLETAR... );
	hipMalloc((void **)&d_B, // COMPLETAR... );
	hipMalloc((void **)&d_C, // COMPLETAR... );
	hipMalloc((void **)&d_R, // COMPLETAR... );
	
	// Init Events
	hipEventCreate(&start);
	hipEventCreate(&stop );
	
	// Start Time Measurement
    hipEventRecord(start, 0);
	
	// Copy Host Data to Device
	
	// COMPLETAR...

	scalarProd<<< // COMPLETAR... >>>(d_C, d_A, d_B, n_elem);
	hipDeviceSynchronize();
	vectorReduce<<< // COMPLETAR...(teniendo en cuenta memoria shared) >>>(d_R, d_C, n_elem);
	
	// Copy Device Data to Host
    
	// COMPLETAR...
	
	// End Time Measurement
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float kernel_time;
	hipEventElapsedTime(&kernel_time, start, stop);
    printf("Execution Time by the GPU: %.2f\n", kernel_time);

	float result_gpu = 0.0f;
	for (int i=0; i < n_block; i++) {
		result_gpu += h_R[i];
	}
	
	// Free Host Memory
	free(h_A); free(h_B); free(h_R);
	
	// Free Device Memory
	hipFree(d_A); hipFree(d_B);
	hipFree(d_C); hipFree(d_R);
	
	// Destroy Events
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	if (result_cpu != result_cpu) {
		printf("Test Failed!\n");
		exit(EXIT_FAILURE);
	}
	printf("Test Passed\n");
	exit(EXIT_SUCCESS);
}
