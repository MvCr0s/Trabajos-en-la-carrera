#include "hip/hip_runtime.h"
%%writefile transpose.cu

/*
 * Matrix Transpose
 *
 * This file contains both device and host code for transposing a matrix.
 *
 */

#include <stdio.h>
 
#define MATRIX_DIM   64
#define SEGMENT_SIZE 32

///////////////////////////////////////////////////////////
//
// Computes the Transpose of a Matrix
//
///////////////////////////////////////////////////////////
__global__ void transposeMatrix(float *d_data, int mat_dim) {

	// Array in Shared Memory
	extern __shared__ float sdata[];
	
	int tid_b = // COMPLETAR...
	int tid_g = // COMPLETAR...
	
	for (int i=0; i < blockDim.x; i++) {
		sdata[// COMPLETAR...] = d_data[// COMPLETAR...];
	}
	
	__syncthreads();
	
	tid_b = // COMPLETAR...
	tid_g = // COMPLETAR...
	
	for (int i=0; i < blockDim.x; i++) {
		d_data[// COMPLETAR...] = sdata[// COMPLETAR...];
	}
}

// ---------------------
// Host Utility Routines
// ---------------------
void transpose(float *At, float *A, const int dim_x, const int dim_y)
{
	for (int y = 0; y < dim_y; y++) {
		for (int x = 0; x < dim_x; x++) {
			At[(x * dim_y) + y] = A[(y * dim_x) + x];
		}
	}
}

bool compareData(float *d_data, float *h_data, int n) {

	for (int i = 0; i < n * n; i++) {
		if (d_data[i] != h_data[i]) {
			return false;
		}
	}
	return true;
}

// ------------
// Main Program
// ------------
int main( void ) {

	// Matrix Dimensions
	int dim_x = MATRIX_DIM;
	int dim_y = dim_x;
	
	// Block Dimension
	int block_dim = SEGMENT_SIZE;
	
	// Number of Blocks
	int n_block = ( dim_x % block_dim == 0 ) // COMPLETAR...
	
	// Execution Configuration Parameters
	dim3 blocksPerGrid  ( // COMPLETAR... );
	dim3 threadsPerBlock( // COMPLETAR... );
	
	// Size (in bytes) Required to Store the Matrix
	size_t n_bytes = (dim_x * dim_y * sizeof(float));
	
	// Allocate Host Memory
	float *A = (float *) malloc(n_bytes);
	float *At = (float *) malloc(n_bytes);
	float *Aux  = (float *) malloc(n_bytes);
	
	// Initialize Host Data
	for (int i = 0; i < (dim_x * dim_y); i++) {
		A[i] = (float) i;
	}
	
	// Compute Reference Transpose Solution
	transpose(At, A, dim_x, dim_y);
	
	// CUDA Events
	hipEvent_t start, stop;
	
	// Performance Data
	float kernel_time, kernel_bandwidth;
	
	// Allocate Device Memory

	// COMPLETAR...

	// Init Events
	hipEventCreate(&start);
	hipEventCreate(&stop );
	
	// Start Time Measurement
    hipEventRecord(start, 0);
	
	// Copy Host Data to Device
	
	// COMPLETAR...	
	
    transposeMatrix<<< // COMPLETAR...(teniendo en cuenta memoria shared) >>>(d_data, dim_x);
	
	// Copy Device Data to Host
	
	// COMPLETAR...
    
	// End Time Measurement
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernel_time, start, stop);

	bool res = compareData(Aux, At, dim_x);
	
	if (res == true) {
		// Report Effective Bandwidth
		kernel_bandwidth = (2.0f * 1000.0f * n_bytes)/(1024 * 1024 * 1024);
		kernel_bandwidth /= kernel_time;
		
		printf( "Throughput = %.4f GB/s, Time = %.5f ms, Size = %u fp32 elements, \n",
				kernel_bandwidth, kernel_time, (dim_x * dim_y) );
	}
	
	// Free Host Memory
	free(A); free(At); free(Aux);
	
	// Free Device Memory
	hipFree(d_data);
	
	// Destroy Events
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	if (res == false) {
		printf("Test Failed!\n");
		exit(EXIT_FAILURE);
	}
	printf("Test Passed\n");
	exit(EXIT_SUCCESS);
}
